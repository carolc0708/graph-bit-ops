
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>

typedef unsigned long long ullong;

// A faster way to obtain lane id in a warp
#define GET_LANEID unsigned laneid; asm("mov.u32 %0, %%laneid;":"=r"(laneid));

//For higher memory access efficiency
template <typename T>
__device__ __inline__ void store64(const void* addr, T a, T b)
{
    *((float2*)addr) = make_float2(*(float*)(&a),*(float*)(&b));
}

//For higher memory access efficiency
template <typename T>
__device__ __inline__ void store128(const void* addr, T a, T b, T c, T d)
{
    *((float4*)addr) = make_float4(*(float*)(&a),*(float*)(&b),*(float*)(&c),*(float*)(&d));
}

// to print unsigned
void bin(unsigned n)
{
    unsigned i;
    for (i = 1 << 31; i > 0; i = i / 2)
        (n & i) ? printf("1") : printf("0");
}

// C = A * A^T => col-major(A) * col-major(A) using rowbyrow model

// col-major packing bit 32
template <typename T>
__global__ void ToBit32Col(const T* __restrict__ A, unsigned* B, const int A_height, const int A_width) // blocksize, nblocks * blocksize
{
    GET_LANEID;
    const unsigned by = blockIdx.y; // nblocks
    const unsigned bx = blockIdx.x; // 1
    unsigned Bval;
#pragma unroll
    for (int i=0; i<32; i++)
    {
        T f0 = A[by*32*32+i*32+laneid];
        unsigned r0 = __brev(__ballot(f0>0));
        if (laneid == i) Bval = r0;
    }
    B[by*32+laneid] = Bval;
}

// row-major packing bit 32
template <typename T>
__global__ void ToBit32Row(const T* __restrict__ A, unsigned* B, const int A_height, const int A_width)
{
    const unsigned bx = blockIdx.x; // blockrows
    const unsigned by = blockIdx.y; // 1
    unsigned Bval=0;
#pragma unroll
    for (int i=0; i<32; i++)
    {
        T f0 = A[bx*32+i];
        Bval = (Bval<<1) + (f0>0);
    }
    B[bx] = Bval;
}

// col-major packing bit 64
template <typename T>
__global__ void ToBit64Col(const T* __restrict__ A, ullong* B, const int A_height, const int A_width)
{
    GET_LANEID;
    const unsigned by = blockIdx.y; //nblocks
    const unsigned bx = blockIdx.x; // 2 <- set this
    ullong Bval;
#pragma unroll
    for (int i=0; i<32; i++)
    {
        T f0 = A[by*64*64+bx*64*32+i*64+laneid]; //
        T f1 = A[by*64*64+bx*64*32+i*64+32+laneid]; //
        unsigned r0 = __ballot(f0>0);
        unsigned r1 = __ballot(f1>0);
        ullong l0;
        asm volatile("mov.b64 %0, {%1,%2};":"=l"(l0):"r"(r0),"r"(r1)); //lo,hi
        if (laneid == i) Bval = __brevll(l0);
    }
    B[by*64+bx*32+laneid] = Bval;
}

// row-major packing bit 64
template <typename T>
__global__ void ToBit64Row(const T* __restrict__  A, ullong* B, const int A_height, const int A_width)
{
    GET_LANEID;
    const unsigned bx = blockIdx.x;
    const unsigned by = blockIdx.y;
    ullong Bval = 0;
#pragma unroll
    for (int i=0; i<64; i++)
    {
        T f0 = A[bx*64+i];
        Bval = (Bval<<1) | (f0>0);
    }
    B[bx] = Bval;
}

// bsr bmm32 no padding
// Cik = Sum(A_ij * B_jk)
// A (bsr matrix) * B (bsr matrix) = C (one float number)
// originally consider to implement C in coo format, but that is not straightforward
// col-bin(32 x (blocksize x nblocks)) x col-bin(32 x (blocksize x nblocks))
template <typename Index, typename T>
__global__ void bmm32_sparse(const unsigned* __restrict__ A, const unsigned* __restrict__ B, T* C,
            const Index* __restrict__ A_rowptr, const Index* __restrict__ A_colind,
            const Index* __restrict__ B_rowptr, const Index* __restrict__ B_colind,
            const Index nblockrows, const Index nblocks)
{
    const int bx = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y + blockIdx.z;
    if (bx < nblockrows) {
        GET_LANEID;
        T* Csub = &C[0];

        // load
        int A_row_start = A_rowptr[bx]; // 0 32 64 . . . 991
        int A_row_end = A_rowptr[bx+1]; // 32 64 96 . . . 991 1022
        const unsigned* Asub = &(A[A_row_start*32]); // block is in continuous layout
        for (int i=A_row_start; i<A_row_end; i++) {
            unsigned r0 = Asub[(i-A_row_start)*32+laneid]; // <--

            int A_col = A_colind[i];
            int B_row_start = B_rowptr[A_col];
            int B_row_end = B_rowptr[A_col+1];
            const unsigned* Bsub = &(B[B_row_start*32]);
            for (int j=B_row_start; j<B_row_end; j++) {
                unsigned r1 = Bsub[(j-B_row_start)*32+laneid]; // <--
                int B_col = B_colind[j];
                register int Cm[32] = {0};

                /* bmm */
                #pragma unroll
                for (int k=0; k<32; k++)
                {
                    unsigned r2 = __shfl(r1, k); //from lane-j, r1 of matrix B

                    // should be protected by critical section !!!
                    Cm[k] += __popc(r0 & r2);
                }
                /* bmm */

                // store
                //C[bx*32+laneid][B_col*32+k] += Cm[k];
                __syncthreads();
                int sum = 0;
                for (int i=0; i<32; i++) sum += Cm[i];
                atomicAdd(Csub+bx, sum);
                __syncthreads();

            } // j in [B_row_start ... B_row_end]
        } // i in [A_row_start ... A_row_end]
    } // if bx < nblockrows + 1
}

// bsr bmv64 no padding
// A (bsr matrix) * B (vector) = C (vector)
template <typename Index, typename T>
__global__ void bmm64_sparse(const ullong* __restrict__ A, const ullong* __restrict__ B,
                            T* C, const int A_height, const int A_width, const int B_width,
                            const Index* __restrict__ rowptr, const Index* __restrict__ colind,
                            const Index nblockrows, const Index nblocks)
{
    const unsigned bx = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y + blockIdx.z;
    if (bx < nblockrows) {
        GET_LANEID;

        // load
        unsigned row_start = rowptr[bx];
        unsigned row_end = rowptr[bx+1];
        const ullong* Asub = &(A[row_start*64]);
        const ullong* Bsub = &(B[0]);
        T* Csub = &(C[bx*64]);
        register unsigned Cm[1] = {0};

        // compute
        for (int i=row_start; i<row_end; i++) {
            Cm[0] = 0;
            ullong a0 = Asub[(i-row_start)*64+laneid];
            ullong a1 = Asub[(i-row_start)*64+32+laneid];
            ullong b0 = Bsub[colind[i]];

            Cm[0] += (__popcll(a0 & b0) << 16) + __popcll(a1 & b0);

            // store
            short t0, t1;
            asm volatile("mov.b32 {%0,%1}, %2;":"=h"(t1),"=h"(t0):"r"(Cm[0]));
            Csub[laneid] += (T)t0;
            Csub[laneid+32] += (T)t1;
        }
    }
}

/**
* Extended Function: tril for csr
* Zeroes out matrix above main diagonal
* Note: for bsr, call it before csr2bsr
*/
// C = tril(A)
template <typename Index, typename T>
__global__ void tril_csr(const Index* A_rowptr, const Index* A_colind, const T* A_csrval,
                         const Index A_nrows, const Index A_nnz,
                         Index* C_rowptr, Index* C_colind, T* C_csrval, Index* C_nnz)
{
    Index remove = 0;
    for (Index row = 0; row < A_nrows; ++row) {
        Index edge_start = A_rowptr[row];
        Index edge_end = A_rowptr[row+1];

        // csrRowPtr_ update must only be done after row loads edge_start
        C_rowptr[row] -= remove;

        for (Index edge = edge_start; edge < edge_end; ++edge) {
            Index col = A_colind[edge];
            if (row < col) {
              remove++;
            } else {
              C_colind[edge-remove] = col;
              C_csrval[edge-remove] = A_csrval[edge];
            }
        }
    }
    // csrRowPtr_ update must be done for last element, which is equivalent to
    // nvals_
    C_rowptr[A_nrows] -= remove;
    C_nnz[0] = A_nnz - remove;
}

/* Extended method, non optimized */
__global__ void reuduceSum(const int *gArr, int arraySize, int *gOut) {
    gOut[0] = 0;
    for (int i=0; i<arraySize; i++) {
        gOut[0] += gArr[i];
    }
}

