
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>

typedef unsigned long long ullong;

// A faster way to obtain lane id in a warp
#define GET_LANEID unsigned laneid; asm("mov.u32 %0, %%laneid;":"=r"(laneid));

//For higher memory access efficiency
template <typename T>
__device__ __inline__ void store64(const void* addr, T a, T b)
{
    *((float2*)addr) = make_float2(*(float*)(&a),*(float*)(&b));
}

//For higher memory access efficiency
template <typename T>
__device__ __inline__ void store128(const void* addr, T a, T b, T c, T d)
{
    *((float4*)addr) = make_float4(*(float*)(&a),*(float*)(&b),*(float*)(&c),*(float*)(&d));
}

// to print unsigned
void bin(unsigned n)
{
    unsigned i;
    for (i = 1 << 31; i > 0; i = i / 2)
        (n & i) ? printf("1") : printf("0");
}

// weight should be col-major packing, layout is 32 * (32*numofblocks)
// input should be row-major packing, layout is whatever it is originally

// col-major packing bit 32
template <typename T>
__global__ void ToBit32Col(const T* __restrict__ A, unsigned* B, const int A_height, const int A_width) // blocksize, nblocks * blocksize
{
    GET_LANEID;
    const unsigned by = blockIdx.y; // nblocks
    const unsigned bx = blockIdx.x; // 1
    unsigned Bval;
#pragma unroll
    for (int i=0; i<32; i++)
    {
        T f0 = A[by*32*32+i*32+laneid];
        unsigned r0 = __brev(__ballot(f0>0));
        if (laneid == i) Bval = r0;
    }
    B[by*32+laneid] = Bval;
}

// row-major packing bit 32
template <typename T>
__global__ void ToBit32Row(const T* __restrict__ A, unsigned* B, const int A_height, const int A_width)
{
    const unsigned bx = blockIdx.x; // blockrows
    const unsigned by = blockIdx.y; // 1
    unsigned Bval=0;
#pragma unroll
    for (int i=0; i<32; i++)
    {
        T f0 = A[bx*32+i];
        Bval = (Bval<<1) + (f0>0);
    }
    B[bx] = Bval;
}

// col-major packing bit 64
template <typename T>
__global__ void ToBit64Col(const T* __restrict__ A, ullong* B, const int A_height, const int A_width)
{
    GET_LANEID;
    const unsigned by = blockIdx.y; //nblocks
    const unsigned bx = blockIdx.x; // 2 <- set this
    ullong Bval;
#pragma unroll
    for (int i=0; i<32; i++)
    {
        T f0 = A[by*64*64+bx*64*32+i*64+laneid]; //
        T f1 = A[by*64*64+bx*64*32+i*64+32+laneid]; //
        unsigned r0 = __ballot(f0>0);
        unsigned r1 = __ballot(f1>0);
        ullong l0;
        asm volatile("mov.b64 %0, {%1,%2};":"=l"(l0):"r"(r0),"r"(r1)); //lo,hi
        if (laneid == i) Bval = __brevll(l0);
    }
    B[by*64+bx*32+laneid] = Bval;
}

// row-major packing bit 64
template <typename T>
__global__ void ToBit64Row(const T* __restrict__  A, ullong* B, const int A_height, const int A_width)
{
    GET_LANEID;
    const unsigned bx = blockIdx.x;
    const unsigned by = blockIdx.y;
    ullong Bval = 0;
#pragma unroll
    for (int i=0; i<64; i++)
    {
        T f0 = A[bx*64+i];
        Bval = (Bval<<1) | (f0>0);
    }
    B[bx] = Bval;
}

// bsr bmm32 no padding
// A (bsr matrix) * B (vector) = C (vector)
// col-bin(32 x (blocksize x nblocks)) * row-bin((nblockrows x nblocks) x 1) = (nblockrow x nblocks) x 1
template <typename Index, typename T>
__global__ void bmm32_sparse(const unsigned* __restrict__ A, const unsigned* __restrict__ B,
            T* C, const int A_height, const int A_width, const int B_width,
            const Index* __restrict__ rowptr, const Index* __restrict__ colind,
            const Index nblockrows, const Index nblocks)
{
    const unsigned bx = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y + blockIdx.z;
    if (bx < nblockrows + 1) {
        GET_LANEID;

        // load
        int row_start = rowptr[bx]; // 0 32 64 . . . 991
        int row_end = rowptr[bx+1]; // 32 64 96 . . . 991 1022

        const unsigned* Asub = &(A[row_start*32]); // block is in continuous layout
        const unsigned* Bsub = &(B[0]); // 0, when it is mv
        T* Csub = &(C[bx*32]);
        register unsigned Cm[1] = {0}; // allocate 1 register

        // compute
        // if that row has more than 1 col block
        for (int i=row_start; i<row_end; i++) {
            Cm[0] = 0;
            unsigned r0 = Asub[(i-row_start)*32+laneid]; // block is in continuous layout
            unsigned r1 = Bsub[(colind[i])]; // only first row is required

            Cm[0] += __popc(r0 & r1);
            // store
            Csub[laneid] += (T)(Cm[0]); //Csub[laneid] = (T)(Cm[0]>0);
        }
    }
}

// bsr bmv64 no padding
// A (bsr matrix) * B (vector) = C (vector)
template <typename Index, typename T>
__global__ void bmm64_sparse(const ullong* __restrict__ A, const ullong* __restrict__ B,
                            T* C, const int A_height, const int A_width, const int B_width,
                            const Index* __restrict__ rowptr, const Index* __restrict__ colind,
                            const Index nblockrows, const Index nblocks)
{
    const unsigned bx = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y + blockIdx.z;
    if (bx < nblockrows + 1) {
        GET_LANEID;

        // load
        unsigned row_start = rowptr[bx];
        unsigned row_end = rowptr[bx+1];
        const ullong* Asub = &(A[row_start*64]);
        const ullong* Bsub = &(B[0]);
        T* Csub = &(C[bx*64]);
        register unsigned Cm[1] = {0};

        // compute
        for (int i=row_start; i<row_end; i++) {
            Cm[0] = 0;
            ullong a0 = Asub[(i-row_start)*64+laneid];
            ullong a1 = Asub[(i-row_start)*64+32+laneid];
            ullong b0 = Bsub[colind[i]];

            Cm[0] += (__popcll(a0 & b0) << 16) + __popcll(a1 & b0);

            // store
            short t0, t1;
            asm volatile("mov.b32 {%0,%1}, %2;":"=h"(t1),"=h"(t0):"r"(Cm[0]));
            Csub[laneid] += (T)t0;
            Csub[laneid+32] += (T)t1;
        }
    }
}

/**
* Extended Function: tril for csr
* Zeroes out matrix above main diagonal
* Note: for bsr, call it before csr2bsr
*/
// C = tril(A)
template <typename Index, typename T>
__global__ void tril_csr(const Index* A_rowptr, const Index* A_colind, const T* A_csrval,
                         const Index A_nrows, const Index A_nnz,
                         Index* C_rowptr, Index* C_colind, T* C_csrval, Index* C_nnz)
{
    Index remove = 0;
    for (Index row = 0; row < A_nrows; ++row) {
        Index edge_start = A_rowptr[row];
        Index edge_end = A_rowptr[row+1];

        // csrRowPtr_ update must only be done after row loads edge_start
        C_rowptr[row] -= remove;

        for (Index edge = edge_start; edge < edge_end; ++edge) {
            Index col = A_colind[edge];
            if (row < col) {
              remove++;
            } else {
              C_colind[edge-remove] = col;
              C_csrval[edge-remove] = A_csrval[edge];
            }
        }
    }
    // csrRowPtr_ update must be done for last element, which is equivalent to
    // nvals_
    C_rowptr[A_nrows] -= remove;
    C_nnz[0] = A_nnz - remove;
}

