#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 1 // do not test more than 1 time for now (vector not clean)
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"

bool check_result(float* p1, float* p2, const int N)
{
    bool flag = true;
    for (int i = 0; i < N; i ++) {

        float diff = p1[i] - p2[i];
        if (fabs(diff) > 1e-6) {
            printf("[%d](%.f,%.f),", i, p1[i], p2[i]);
            flag = false;
        }
    }
    return flag;
}

bool check_result(float* p1, int* p2, const int N)
{
    bool flag = true;
    for (int i = 0; i < N * N; i ++) {
        //printf("(%.0f,%d)",p1[i],p2[i]);
        float diff = p1[i] - (float)p2[i];
        if (fabs(diff) > 1e-6) {
            flag = false;
        }
    }
    return flag;
}

int countnnzinvec(const float* vec, const int N)
{
    int counter = 0;
    for (int i=0; i<N; i++) if (vec[i] != 0) counter += 1;
    return counter;
}

void printvec(float* vec, const int N)
{
    for(int i=0; i<N; i++) printf(vec[i]>0?"1":"0");
    printf("\n");
}

void printresvec(float* vec, const int N)
{
    for(int i=0; i<N; i++) printf("%d", (int)vec[i]);
    printf("\n");
}

void printmat(float* bsrval, const int nblocks, const int blocksize)
{
    for(int i=0; i<nblocks; i++) {
        printf("[%d]\n", i);
        for(int j=0; j<blocksize; j++) {
            for(int k=0; k<blocksize; k++) {
                printf(bsrval[i*blocksize*blocksize+j*blocksize+k]>0?"1":"0");
            }
            printf("\n");
        }
        printf("\n");
    }
}

void printbinvec(unsigned* binvec, const int N)
{
    for(int i=0; i<N; i++) bin(binvec[i]);
    printf("\n");
}

void printbinmat(unsigned* binbsrval, const int nblocks, const int blocksize)
{
    for(int i=0; i<nblocks; i++) {
        printf("[%d]\n", i);
        for(int j=0; j<blocksize; j++) {
            bin(binbsrval[i*blocksize+j]);
            printf("\n");
        }
        printf("\n");
    }
}

__global__ void printpackvec (ullong* packvec, const int N)
{
    for(int i=0; i<N; i++) {
        ullong j;
        for(j = 1ULL << 63; j > 0; j = j / 2)
            (packvec[i] & j) ? printf("1") : printf("0");
    }
    printf("\n");
}

__global__ void printpackmat (ullong* packbsrval,  const int nblocks, const int blocksize)
{
    for(int i=0; i<nblocks; i++) {
        printf("[%d]\n", i);
        for(int j=0; j<blocksize; j++) {
            ullong k;
            for(k = 1ULL << 63; k > 0; k = k / 2)
                (packbsrval[i*blocksize+j] & k) ? printf("1") : printf("0");

            printf("\n");
        }
        printf("\n");
    }
}

void printind(int* indarr, const int N)
{
    for(int i=0; i<N; i++) {
        printf("%d ", indarr[i]);
    }
    printf("\n");
}

int main32(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // matrix storage -----------------------------------
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_COLUMN;

    // csr2bsr in column-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);

    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));
    csr2bsr_batch_32(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks);

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* dummy_B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     dummy_B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_SYMBOLIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipFree(B_cscRowInd);
    hipFree(B_cscColPtr);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    unsigned* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(unsigned));
    csr2bsr_batch_32(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks);
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);

//    printDeviceIndArr<<<1,1>>>(A_bsrRowPtr, (nblockrows+1));
//    printDeviceIndArr<<<1,1>>>(A_bsrColInd, nblocks);
//    printDeviceIndArr<<<1,1>>>(B_bsrRowPtr, (nblockrows+1));
//    printDeviceIndArr<<<1,1>>>(B_bsrColInd, nblocks);

	// time measurement setup -----------------------------------
	hipEvent_t start, stop;
	float milliseconds = 0;

    // ============================================= BSTC-32 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * nblockrows);
	setDeviceValArr<<<1,1>>>(fC, nblockrows, 0);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // get grid dim
    double nbr = cbrt((double)nblockrows);
    int blockdim = (int)ceil(nbr);
    printf("cbrt(nblockrows) = %d\n", blockdim);
    dim3 grid(blockdim, blockdim, blockdim);

    // ------
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmm32_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
                                               A_bsrRowPtr, A_bsrColInd,
                                               B_bsrRowPtr, B_bsrColInd,
                                               nblockrows, nblocks);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm32_time = (milliseconds*1e3)/double(TEST_TIMES);

    // ------

//    printf("fC: \n"); printDeviceIndArr<<<1,1>>>(fC, nblockrows);
    int* result_bsrbmm32;
    hipMalloc((void**)&result_bsrbmm32, sizeof(int) * 1);
    reuduceSum<<<1,1>>>(fC, nblockrows, result_bsrbmm32);
    int ntris;
    hipMemcpy(&ntris, result_bsrbmm32, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris: %d\n", ntris);
    hipFree(result_bsrbmm32);


    // ============================================= cuSPARSE csr spmm-float
//    // setup cusparse metadata
//    hipsparseHandle_t handle_csr;
//    hipsparseCreate(&handle_csr);
//
//    hipsparseMatDescr_t A_descr;
//    hipsparseCreateMatDescr(&A_descr);
//    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//    hipsparseMatDescr_t B_descr;
//    hipsparseCreateMatDescr(&B_descr);
//    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//    hipsparseMatDescr_t C_descr;
//    hipsparseCreateMatDescr(&C_descr);
//    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);
//
//    // setup input and output csr storage
//    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
//    // duplicate A as B
//    int* B_csrRowPtr, *B_csrColInd;
//    float* B_csrVal;
//    int B_nnz = A_nnz;
//    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
//    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
//    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
//    hipMemcpy(B_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
//    hipMemcpy(B_csrColInd, A_csrColInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
//    hipMemcpy(B_csrVal, A_csrVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);
//
//    // calculate nnz in C and allocate storage
//    int* C_csrRowPtr, *C_csrColInd;
//    float* C_csrVal;
//    int C_nnz;
//    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
//    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, nrows, nrows, nrows,
//                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
//                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
//                        C_descr, C_csrRowPtr, &C_nnz);
//    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
//    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);
//    printf("result C nnz: %d\n", C_nnz);
//
//    //
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//
//    // ------
//    hipEventRecord(start);
//    for (int i=0; i<TEST_TIMES; i++) {
//        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, nrows, nrows, nrows,
//                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
//                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
//                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
//    }
//
//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//
//    milliseconds = 0;
//    hipEventElapsedTime(&milliseconds,start,stop);
//    double cusparsecsrspgemmfloat_time = (milliseconds*1e3)/double(TEST_TIMES);
//    // ------
//
//    // the result include csrvalC, C_csrRowPtr, C_csrColInd
//    //float* result_cusparsecsrspgemmfloat = (float*)malloc(ncols * 1 * sizeof(float));
//    //hipMemcpy(result_cusparsecsrspgemmfloat, dY, ncols * 1 * sizeof(float), hipMemcpyDeviceToHost);
//

    //============================================= CHECK RESULT
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-32 success: %d\n", check_result(result_bsrbmv32, result_cusparsebsrspmvfloat, ncols));

    printf("BSR BMM-32: %.3lf\n", bmm32_time);
//    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
//    hipFree(C_csrVal);
//    hipFree(C_csrColInd);
//    hipFree(C_csrRowPtr);
//    hipFree(B_csrVal);
//    hipFree(B_csrColInd);
//    hipFree(B_csrRowPtr);
//    hipFree(d_A_nnz);
//    hipFree(A_csrVal);
//    hipFree(A_csrColInd);
//    hipFree(A_csrRowPtr);
//    hipsparseDestroyMatDescr(C_descr);
//    hipsparseDestroyMatDescr(B_descr);
//    hipsparseDestroyMatDescr(A_descr);
//    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

int main(int argc, char* argv[])
{
    main32(argc, argv);
    //main64(argc, argv);
}
