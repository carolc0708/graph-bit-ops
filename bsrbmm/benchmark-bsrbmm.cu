#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 1
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"

//======================================================================================
// bsrbmm4
//======================================================================================
int main4(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now
    unsigned csrbytes = (nrows+1+A_nnz*2) * 4 * 2;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 4;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 1 + (nblockrows+1+nblocks) * 4) * 2;
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    uchar* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(uchar));
    csr2bsr_batch_4_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    uchar* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(uchar));
    csr2bsr_batch_4_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);


    // ============================================= BSTC-4 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * 1);
	setDeviceValArr<int, int><<<1,1>>>(fC, 1, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);
#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

//        bmm4_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
//                                               A_bsrRowPtr, A_bsrColInd,
//                                               B_bsrRowPtr, B_bsrColInd,
//                                               nblockrows, nblocks, nrows);
        bmm4_sparse_masked_v4<int, int><<<grid, 32>>>(tA, tB, fC,
                                                   A_bsrRowPtr, A_bsrColInd,
                                                   B_bsrRowPtr, B_bsrColInd,
                                                   nblockrows, nblocks, nrows);
    }

    bmm_timer.Stop();
    double bmm4_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    int ntris_bmm;
    hipMemcpy(&ntris_bmm, fC, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-4: %.3lf\n", bmm4_time);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    setDeviceValArr<int, int><<<1,1>>>(C_csrColInd, C_nnz, 0);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);
    setDeviceValArr<int, float><<<1,1>>>(C_csrVal, C_nnz, 0);
//    printf("result C_csrVal nnz: %d\n", C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {

//       bmm4_sparse_full<int, float><<<grid, 32>>>(tA, tB, C_csrVal,
//                                            A_bsrRowPtr, A_bsrColInd,
//                                            B_bsrRowPtr, B_bsrColInd,
//                                            C_csrRowPtr, C_csrColInd,
//                                            nblockrows, nblocks, nrows);

        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

//    printResVec<<<1,1>>>(C_csrVal, C_nnz);

    // ------

    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-4 success: %d\n", check_result(result_bsrbmv4, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}


//======================================================================================
// bsrbmm8
//======================================================================================
int main8(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now
    unsigned csrbytes = (nrows+1+A_nnz*2) * 4 * 2;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 8;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 1 + (nblockrows+1+nblocks) * 4) * 2;
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    uchar* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(uchar));
    csr2bsr_batch_8_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    uchar* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(uchar));
    csr2bsr_batch_8_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);


    // ============================================= BSTC-8 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * 1);
	setDeviceValArr<int, int><<<1,1>>>(fC, 1, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);
#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmm8_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
                                               A_bsrRowPtr, A_bsrColInd,
                                               B_bsrRowPtr, B_bsrColInd,
                                               nblockrows, nblocks, nrows, runtime);
    }

    bmm_timer.Stop();
    double bmm8_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------
#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif

    int ntris_bmm;
    hipMemcpy(&ntris_bmm, fC, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-8: %.3lf\n", bmm8_time);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-8 success: %d\n", check_result(result_bsrbmv8, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

//======================================================================================
// bsrbmm16
//======================================================================================
int main16(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now
    unsigned csrbytes = (nrows+1+A_nnz*2) * 4 * 2;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 16;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 2 + (nblockrows+1+nblocks) * 4) * 2;
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    ushort* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(ushort));
    csr2bsr_batch_16_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    ushort* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(ushort));
    csr2bsr_batch_16_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);

    // ============================================= BSTC-16 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * 1);
	setDeviceValArr<int, int><<<1,1>>>(fC, 1, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);
#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmm16_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
                                               A_bsrRowPtr, A_bsrColInd,
                                               B_bsrRowPtr, B_bsrColInd,
                                               nblockrows, nblocks, nrows, runtime);
    }

    bmm_timer.Stop();
    double bmm16_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------
#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif


    int ntris_bmm;
    hipMemcpy(&ntris_bmm, fC, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-16: %.3lf\n", bmm16_time);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-16 success: %d\n", check_result(result_bsrbmv16, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

//======================================================================================
// bsrbmm32
//======================================================================================
int main32(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now
    unsigned csrbytes = (nrows+1+A_nnz*2) * 4 * 2;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 4 + (nblockrows+1+nblocks) * 4) * 2;
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));
    csr2bsr_batch_32_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    unsigned* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(unsigned));
    csr2bsr_batch_32_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);

    // ============================================= BSTC-32 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * 1);
	setDeviceValArr<int, int><<<1,1>>>(fC, 1, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);

#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

//        bmm32_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
//                                           A_bsrRowPtr, A_bsrColInd,
//                                           B_bsrRowPtr, B_bsrColInd,
//                                           nblockrows, nblocks, nrows, runtime);

//        bmm32_sparse_masked<int, int><<<grid, 32>>>(tA, tB, fC,
//                                                   A_bsrRowPtr, A_bsrColInd,
//                                                   B_bsrRowPtr, B_bsrColInd,
//                                                   nblockrows, nblocks, nrows);

//        bmm32_sparse_masked_v3<int, int><<<grid, 32>>>(tA, tB, fC,
//                                                    A_bsrRowPtr, A_bsrColInd,
//                                                    B_bsrRowPtr, B_bsrColInd,
//                                                    nblockrows, nblocks, nrows);

        bmm32_sparse_masked_v4<int, int><<<grid, 32>>>(tA, tB, fC,
                                                    A_bsrRowPtr, A_bsrColInd,
                                                    B_bsrRowPtr, B_bsrColInd,
                                                    nblockrows, nblocks, nrows);
    }

    bmm_timer.Stop();
    double bmm32_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------
#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif

    int ntris_bmm;
    hipMemcpy(&ntris_bmm, fC, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-32: %.3lf\n", bmm32_time);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    setDeviceValArr<int, int><<<1,1>>>(C_csrColInd, C_nnz, 0);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);
    setDeviceValArr<int, float><<<1,1>>>(C_csrVal, C_nnz, 0);
//    printf("result C_csrVal nnz: %d\n", C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {


//       bmm32_sparse_full_v1<int, float><<<grid, 32>>>(tA, tB, C_csrVal,
//                                                A_bsrRowPtr, A_bsrColInd,
//                                                B_bsrRowPtr, B_bsrColInd,
//                                                C_csrRowPtr, C_csrColInd,
//                                                nblockrows, nblocks, nrows);

        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

//    printResVec<<<1,1>>>(C_csrVal, C_nnz);
//    printResVec<<<1,1>>>(A_csrVal, A_nnz);

    // ------

    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-32 success: %d\n", check_result(result_bsrbmv32, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

//======================================================================================
// bsrbmm64
//======================================================================================
int main64(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now
    unsigned csrbytes = (nrows+1+A_nnz*2) * 4 * 2;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 64;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 8 + (nblockrows+1+nblocks) * 4) * 2;
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    ullong* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(ullong));
    csr2bsr_batch_64_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    ullong* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(ullong));
    csr2bsr_batch_64_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);

    // ============================================= BSTC-64 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * 1);
	setDeviceValArr<int, int><<<1,1>>>(fC, 1, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);

#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmm64_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
                                               A_bsrRowPtr, A_bsrColInd,
                                               B_bsrRowPtr, B_bsrColInd,
                                               nblockrows, nblocks, nrows, runtime);
    }

    bmm_timer.Stop();
    double bmm64_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------
#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif

    int ntris_bmm;
    hipMemcpy(&ntris_bmm, fC, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-64: %.3lf\n", bmm64_time);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    // the result include C_csrVal, C_csrRowPtr, C_csrColInd
    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-32 success: %d\n", check_result(result_bsrbmv32, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

int main(int argc, char* argv[])
{
#if BLOCKSIZE == 64
    main64(argc, argv);
#elif BLOCKSIZE == 32
    main32(argc, argv);
#elif BLOCKSIZE == 16
    main16(argc, argv);
#elif BLOCKSIZE == 8
    main8(argc, argv);
#else
    main4(argc, argv);
#endif
}
