#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 1
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"

//======================================================================================
// bsrbmm32
//======================================================================================
int main32(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now

//    printDeviceIndArr<int><<<1,1>>>(A_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(A_csrColInd, A_nnz);
//    printResVec<float><<<1,1>>>(A_csrVal, A_nnz);

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));
    csr2bsr_batch_32_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
//    hipFree(B_cscRowInd);
//    hipFree(B_cscColPtr);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    unsigned* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(unsigned));
    csr2bsr_batch_32_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);

//    printDeviceIndArr<int><<<1,1>>>(A_bsrRowPtr, nblockrows+1);
//    printDeviceIndArr<int><<<1,1>>>(A_bsrColInd, nblocks);
//    printBin32Vec<<<1,1>>>(tA, nblocks*blocksize);
//    printDeviceIndArr<int><<<1,1>>>(B_bsrRowPtr, nblockrows+1);
//    printDeviceIndArr<int><<<1,1>>>(B_bsrColInd, nblocks);
//    printBin32Vec<<<1,1>>>(tB, nblocks*blocksize);


    // ============================================= BSTC-32 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * nblockrows);
	setDeviceValArr<int, int><<<1,1>>>(fC, nblockrows, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);
    printf("cbrt(nblockrows) = %d\n", gridDim);

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmm32_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
                                               A_bsrRowPtr, A_bsrColInd,
                                               B_bsrRowPtr, B_bsrColInd,
                                               nblockrows, nblocks, nrows, runtime);
    }

    bmm_timer.Stop();
    double bmm32_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------
#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif

//    printf("fC: \n"); printResVec<int><<<1,1>>>(fC, nblockrows);
    int* result_bsrbmm32;
    hipMalloc((void**)&result_bsrbmm32, sizeof(int) * 1);
    reuduceSum<int><<<1,1>>>(fC, nblockrows, result_bsrbmm32);
    int ntris_bmm;
    hipMemcpy(&ntris_bmm, result_bsrbmm32, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("==========================================\n");
    unsigned size = (nblocks * blocksize * 4 + (nblockrows+1+nblocks) * 4) * 2/1024;
    printf("nblocks: %d, bsr total size: %d (KB)\n", nblocks, (int)size);
    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-32: %.3lf\n", bmm32_time);

    hipFree(result_bsrbmm32);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

//    printDeviceIndArr<int><<<1,1>>>(A_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(A_csrColInd, A_nnz);
//    printDeviceIndArr<int><<<1,1>>>(B_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(B_csrColInd, B_nnz);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);
//    printf("result C_csrVal nnz: %d\n", C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

//    printDeviceIndArr<int><<<1,1>>>(C_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(C_csrColInd, C_nnz);
//    printResVec<float><<<1,1>>>(C_csrVal, C_nnz);

    // ------

    // the result include C_csrVal, C_csrRowPtr, C_csrColInd
//    printResVec<int><<<1,1>>>(C_csrRowPtr, nrows+1);
//    printResVec<int><<<1,1>>>(C_csrColInd, C_nnz);
//    printf("C_csrVal: \n"); printResVec<float><<<1,1>>>(C_csrVal, C_nnz);

//    float* resvec;
//    hipMalloc((void**)&resvec, sizeof(float) * nblockrows);
//    setDeviceValArr<int, float><<<1,1>>>(resvec, nblockrows, 0.0);
//    gatherNnzbyBlockrow<<<1,1>>>(C_csrRowPtr, C_csrColInd, C_csrVal,
//                                 nrows, nblockrows, blocksize, resvec);
//    printf("C_csrVal (gather by blockrow): \n"); printResVec<float><<<1,1>>>(resvec, nblockrows);


    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-32 success: %d\n", check_result(result_bsrbmv32, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

//======================================================================================
// bsrbmm64
//======================================================================================
int main64(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);
//    removeDiagonalNnz<<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows);

    // process input matrix to simulate tc algorithm
    // C = A * A^T
    // 1) get A = graphblas::tril(A)
    // duplicate matrix as A
    int* A_csrRowPtr, *A_csrColInd;
    float* A_csrVal;
    int* d_A_nnz;
    int A_nnz;
    hipMalloc(&A_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&A_csrColInd, sizeof(int) * nnz);
    hipMalloc(&A_csrVal, sizeof(float) * nnz);
    hipMalloc(&d_A_nnz, sizeof(int) * 1);
    hipMemcpy(A_csrRowPtr, csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrColInd, csrColInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(A_csrVal, csrVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    // call tril()
    tril_csr<int, float><<<1,1>>>(csrRowPtr, csrColInd, csrVal, nrows, nnz,
                                  A_csrRowPtr, A_csrColInd, A_csrVal, d_A_nnz);
    hipMemcpy(&A_nnz, d_A_nnz, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("nnz before tril_csr(): %d, after tril_csr(): %d\n", nnz, A_nnz); // <- we ignore A's [A_nnz to nnz] from now

//    printDeviceIndArr<int><<<1,1>>>(A_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(A_csrColInd, A_nnz);
//    printResVec<float><<<1,1>>>(A_csrVal, A_nnz);

    // reset host csr with updated matrix
    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * A_nnz);
    h_csrVal = (float*) malloc(sizeof(float) * A_nnz);
    hipMemcpy(h_csrRowPtr, A_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColInd, A_csrColInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrVal, A_csrVal, sizeof(float) * A_nnz, hipMemcpyDeviceToHost);

	// transform from csr to bsr using cuSPARSE
	int* A_bsrRowPtr, *A_bsrColInd;
	float* A_bsrVal;
	int blocksize = 64;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&A_bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        A_csrRowPtr, A_csrColInd, blocksize, bsr_descr, A_bsrRowPtr, &nblocks);
    hipMalloc((void**)&A_bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);

    // free cusparse descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // packed matrix tA
    ullong* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(ullong));
    csr2bsr_batch_64_row(h_csrRowPtr, h_csrColInd, nrows, ncols, A_nnz,
                     A_bsrRowPtr, A_bsrColInd, tA, blocksize, nblockrows, nblocks); //row-major

    // csr2csc for B as A^T
    int* B_cscRowInd, *B_cscColPtr;
    float* B_cscVal;
    hipMalloc(&B_cscRowInd, sizeof(int) * A_nnz);
    hipMalloc(&B_cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_cscVal, sizeof(float) * A_nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, A_nnz,
                     A_csrVal, A_csrRowPtr, A_csrColInd,
                     B_cscVal, B_cscRowInd, B_cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_B_cscRowInd, *h_B_cscColPtr;
    h_B_cscRowInd = (int*) malloc(sizeof(int) * A_nnz);
    h_B_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_B_cscRowInd, B_cscRowInd, sizeof(int) * A_nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_B_cscColPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);
//    hipFree(B_cscRowInd);
//    hipFree(B_cscColPtr);

    // csr2bsr for B & pack matrix for tB
    int* B_bsrRowPtr, *B_bsrColInd;
    hipMalloc(&B_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&B_bsrColInd, sizeof(int) * nblocks);
    ullong* tB;
    hipMalloc((void**)&tB, nblocks * blocksize * sizeof(ullong));
    csr2bsr_batch_64_col(h_B_cscColPtr, h_B_cscRowInd, nrows, ncols, A_nnz,
                     B_bsrRowPtr, B_bsrColInd, tB, blocksize, nblockrows, nblocks); //col-major
    free(h_B_cscRowInd);
    free(h_B_cscColPtr);

//    printDeviceIndArr<int><<<1,1>>>(A_bsrRowPtr, nblockrows+1);
//    printDeviceIndArr<int><<<1,1>>>(A_bsrColInd, nblocks);
//    printBin32Vec<<<1,1>>>(tA, nblocks*blocksize);
//    printDeviceIndArr<int><<<1,1>>>(B_bsrRowPtr, nblockrows+1);
//    printDeviceIndArr<int><<<1,1>>>(B_bsrColInd, nblocks);
//    printBin32Vec<<<1,1>>>(tB, nblocks*blocksize);


    // ============================================= BSTC-64 bsr bmm
    // allocate bsr storage for resulting C
    // use 1 float to store the reduced sum for now
    int* fC;
	hipMalloc((void**)&fC, sizeof(int) * nblockrows);
	setDeviceValArr<int, int><<<1,1>>>(fC, nblockrows, 0);

    // get grid dim
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);
    printf("cbrt(nblockrows) = %d\n", gridDim);

int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmm_timer;
    bmm_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmm64_sparse<int, int><<<grid, 32>>>(tA, tB, fC,
                                               A_bsrRowPtr, A_bsrColInd,
                                               B_bsrRowPtr, B_bsrColInd,
                                               nblockrows, nblocks, nrows, runtime);
    }

    bmm_timer.Stop();
    double bmm64_time = bmm_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------
#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif

//    printf("fC: \n"); printResVec<int><<<1,1>>>(fC, nblockrows);
    int* result_bsrbmm64;
    hipMalloc((void**)&result_bsrbmm64, sizeof(int) * 1);
    reuduceSum<int><<<1,1>>>(fC, nblockrows, result_bsrbmm64);
    int ntris_bmm;
    hipMemcpy(&ntris_bmm, result_bsrbmm64, sizeof(int) * 1, hipMemcpyDeviceToHost);


    printf("==========================================\n");
    unsigned size = (nblocks * blocksize * 8 + (nblockrows+1+nblocks) * 4) * 2/1024;
    printf("nblocks: %d, bsr total size: %d (KB)\n", nblocks, (int)size);
    printf("ntris_bmm: %d\n", ntris_bmm);
    printf("BSR BMM-64: %.3lf\n", bmm64_time);

    hipFree(result_bsrbmm64);

    // ============================================= cuSPARSE csr spgemm-float
    // setup cusparse metadata
    hipsparseHandle_t handle_csr;
    hipsparseCreate(&handle_csr);

    hipsparseMatDescr_t A_descr;
    hipsparseCreateMatDescr(&A_descr);
    hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t B_descr;
    hipsparseCreateMatDescr(&B_descr);
    hipsparseSetMatType(B_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(B_descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t C_descr;
    hipsparseCreateMatDescr(&C_descr);
    hipsparseSetMatType(C_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(C_descr, HIPSPARSE_INDEX_BASE_ZERO);

    // setup input and output csr storage
    // 2) get B = A^T (get B = A here, and transpose using HIPSPARSE_OPERATION_TRANSPOSE)
    // duplicate A as B
    int* B_csrRowPtr, *B_csrColInd;
    float* B_csrVal;
    int B_nnz = A_nnz;
    hipMalloc(&B_csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&B_csrColInd, sizeof(int) * B_nnz);
    hipMalloc(&B_csrVal, sizeof(float) * B_nnz);
    hipMemcpy(B_csrRowPtr, B_cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrColInd, B_cscRowInd, sizeof(int) * B_nnz, hipMemcpyDeviceToDevice);
    hipMemcpy(B_csrVal, B_cscVal, sizeof(float) * B_nnz, hipMemcpyDeviceToDevice);

//    printDeviceIndArr<int><<<1,1>>>(A_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(A_csrColInd, A_nnz);
//    printDeviceIndArr<int><<<1,1>>>(B_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(B_csrColInd, B_nnz);

    // calculate nnz in C and allocate storage
    int* C_csrRowPtr, *C_csrColInd;
    float* C_csrVal;
    int C_nnz;
    hipMalloc(&C_csrRowPtr, sizeof(int) * (nrows+1));
    hipsparseXcsrgemmNnz(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                        A_descr, A_nnz, A_csrRowPtr, A_csrColInd,
                        B_descr, B_nnz, B_csrRowPtr, B_csrColInd,
                        C_descr, C_csrRowPtr, &C_nnz);

    hipMalloc(&C_csrColInd, sizeof(int) * C_nnz);
    hipMalloc(&C_csrVal, sizeof(float) * C_nnz);
//    printf("result C_csrVal nnz: %d\n", C_nnz);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrgemm(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, nrows, nrows,
                         A_descr, A_nnz, A_csrVal, A_csrRowPtr, A_csrColInd,
                         B_descr, B_nnz, B_csrVal, B_csrRowPtr, B_csrColInd,
                         C_descr, C_csrVal, C_csrRowPtr, C_csrColInd);
    }

    csr_timer.Stop();
    double cusparsecsrspgemmfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

//    printDeviceIndArr<int><<<1,1>>>(C_csrRowPtr, nrows+1);
//    printDeviceIndArr<int><<<1,1>>>(C_csrColInd, C_nnz);
//    printResVec<float><<<1,1>>>(C_csrVal, C_nnz);

    // ------

    // the result include C_csrVal, C_csrRowPtr, C_csrColInd
    int* result_cusparsecsrspgemmfloat;
    hipMalloc((void**)&result_cusparsecsrspgemmfloat, sizeof(int) * 1);
    reuduceSum<float><<<1,1>>>(C_csrVal, C_nnz, result_cusparsecsrspgemmfloat);
    int ntris_csr;
    hipMemcpy(&ntris_csr, result_cusparsecsrspgemmfloat, sizeof(int) * 1, hipMemcpyDeviceToHost);
    printf("ntris_csr: %d\n", ntris_csr);


    //============================================= check result
    //printf("CuSPARSE CSR SpGEMM-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMM-32 success: %d\n", check_result(result_bsrbmv32, result_cusparsebsrspmvfloat, ncols));

    printf("CuSPARSE CSR SpGEMM-float: %.3lf\n", cusparsecsrspgemmfloat_time);

    //============================================= free memory
    // free bsr bmm
    hipFree(fC);

    hipFree(tB);
    hipFree(B_bsrColInd);
    hipFree(B_bsrRowPtr);

    hipFree(tA);
    hipFree(A_bsrColInd);
    hipFree(A_bsrRowPtr);


    // free cusparse csr spmv
    hipFree(C_csrVal);
    hipFree(C_csrColInd);
    hipFree(C_csrRowPtr);
    hipFree(B_csrVal);
    hipFree(B_csrColInd);
    hipFree(B_csrRowPtr);
    hipFree(d_A_nnz);
    hipFree(A_csrVal);
    hipFree(A_csrColInd);
    hipFree(A_csrRowPtr);
    hipsparseDestroyMatDescr(C_descr);
    hipsparseDestroyMatDescr(B_descr);
    hipsparseDestroyMatDescr(A_descr);
    hipsparseDestroy(handle_csr);

    // free mem
    hipFree(csrVal);
    hipFree(csrColInd);
    hipFree(csrRowPtr);

    free(h_csrVal);
    free(h_csrColInd);
    free(h_csrRowPtr);

    // free all results
}

int main(int argc, char* argv[])
{
#if BLOCKSIZE == 64
    main64(argc, argv);
#else
    main32(argc, argv);
#endif
}
