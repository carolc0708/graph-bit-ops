#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 5
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"
#include "eval-func.cu"

/* bsrbmv-4 */
int main4(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 4;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 1 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    uchar* tA;
    hipMalloc((void**)&tA, ceil((float)nblocks/64) * 64 * blocksize * sizeof(uchar));

    // use batch transform as default
    csr2bsr_batch_4(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
//    printGlobalBSRBlock4<<<1,1>>>(tA, blocksize, 5);

     printBlockReport<int, uchar><<<1,1>>>(tA, bsrRowPtr, bsrColInd, nblockrows, nblocks, blocksize);

     hipFree(tA);
     hipFree(bsrRowPtr);
     hipFree(bsrColInd);
}

/* bsrbmv-8 */
int main8(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 8;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 1 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    uchar* tA;
    hipMalloc((void**)&tA, ceil((float)nblocks/16) * 16 * blocksize * sizeof(uchar));

#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit8Col<float><<<dim3(1, ceil((float)temp_nblocks/16)), 32>>>(bsrVal, tA, nblocks); // sparse matrix
    //printGlobalBSRBlock8<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_8(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
//    printGlobalBSRBlock8<<<1,1>>>(tA, blocksize, 5);
#endif

     printBlockReport<int, uchar><<<1,1>>>(tA, bsrRowPtr, bsrColInd, nblockrows, nblocks, blocksize);

     hipFree(tA);
     hipFree(bsrRowPtr);
     hipFree(bsrColInd);
}

/* bsrbmv-16 */
int main16(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 16;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 2 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    ushort* tA;
    hipMalloc((void**)&tA, ceil((float)nblocks/4) * 4 * blocksize * sizeof(ushort)); // <--

#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit16Col<float><<<dim3(1, ceil((float)temp_nblocks/4)), 32>>>(bsrVal, tA, nblocks); // sparse matrix
    //printGlobalBSRBlock16<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_16(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
//    printGlobalBSRBlock16<<<1,1>>>(tA, blocksize, 5);
#endif

     printBlockReport<int, ushort><<<1,1>>>(tA, bsrRowPtr, bsrColInd, nblockrows, nblocks, blocksize);

     hipFree(tA);
     hipFree(bsrRowPtr);
     hipFree(bsrColInd);
}

/* bsrbmv-32 */
int main32(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 4 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));

#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit32Col<float><<<dim3(1, nblocks), 32>>>(bsrVal, tA, blocksize, nblocks * blocksize); // sparse matrix
//    printGlobalBSRBlock32<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_32(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
#endif

     printBlockReport<int, unsigned><<<1,1>>>(tA, bsrRowPtr, bsrColInd, nblockrows, nblocks, blocksize);

     hipFree(tA);
     hipFree(bsrRowPtr);
     hipFree(bsrColInd);
}

/* bsrbmv-64 */
int main64(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 64;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate nblocks first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 8 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix tA
    ullong* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(ullong));


#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit64Col<float><<<dim3(1, nblocks), 32>>>(bsrVal, tA, blocksize, nblocks * blocksize); // sparse matrix
//    printGlobalBSRBlock64<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_64(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
#endif

     printBlockReportUllong<int, ullong><<<1,1>>>(tA, bsrRowPtr, bsrColInd, nblockrows, nblocks, blocksize);

     hipFree(tA);
     hipFree(bsrRowPtr);
     hipFree(bsrColInd);
}

int main(int argc, char* argv[])
{
#if BLOCKSIZE == 64
    main64(argc, argv);
#elif BLOCKSIZE == 32
    main32(argc, argv);
#elif BLOCKSIZE == 16
    main16(argc, argv);
#elif BLOCKSIZE == 8
    main8(argc, argv);
#else
    main4(argc, argv);
#endif
}
