#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 1
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"

/* bsrbmv-8 */
int main8(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 8;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 1 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    uchar* tA;
    hipMalloc((void**)&tA, ceil((float)nblocks/16) * 16 * blocksize * sizeof(uchar));

#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit8Col<float><<<dim3(1, ceil((float)temp_nblocks/16)), 32>>>(bsrVal, tA, nblocks); // sparse matrix
    //printGlobalBSRBlock8<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_8(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
//    printGlobalBSRBlock8<<<1,1>>>(tA, blocksize, 5);
#endif

    // ============================================= input vector storage
    // generate random vector
    srand(time(0));
	float *B = (float*)malloc((nblockrows * blocksize) * 1 * sizeof(float));
	for (int i = 0; i < (nblockrows * blocksize) * 1; i ++)
    {
        float x = (float)rand() / RAND_MAX;
        if (i >= ncols) B[i] = 0;
        else B[i] = (x > 0.5) ? 1 : 0;
    }

#ifdef VERBOSE
    printf("initialize a vector with size %d x 1\n", (nblockrows * blocksize));
//    printf("orivec: \n"); printHostVec(B, (nblockrows * blocksize));
#endif

    // copy to device
	float *fB;
	hipMalloc(&fB, (nblockrows * blocksize) * 1 * sizeof(float));
	hipMemcpy(fB, B, (nblockrows * blocksize) * 1 * sizeof(float), hipMemcpyHostToDevice);

    // pack B
    uchar *tB;
    hipMalloc(&tB, ceil((float)nblockrows/4)* 4 * sizeof(uchar));
    setDeviceValArr<int, uchar><<<1,1>>>(tB, ceil((float)nblockrows/4)*4, 0);

    // get gridDim, this is to avoid nblockrows being larger than MAX_gridDim (65535?!)
    int gridDim = (int)ceil(cbrt((double)nblockrows/4));
    dim3 grid(gridDim, gridDim, gridDim);

#ifdef VERBOSE
    printf("ceil(nblockrows/4) = %d, cbrt(nblockrows/4) = %d\n", (int)ceil((double)nblockrows/4), gridDim);
#endif

    ToBit8Row<float><<<grid, 32>>>(fB, tB, nblockrows); // dense vector
//    printf("binarized vec: \n"); printBin8Vec<<<1,1>>>(tB, nblockrows);


    // ============================================= BSTC-8 bsr bmv
    // init C (result storage)
    float *fC;
    hipMalloc(&fC, (nblockrows * blocksize) * 1 * sizeof(float));
    setDeviceValArr<int, float><<<1,1>>>(fC, nblockrows * blocksize, 0);

    int gridDim_new = (int)ceil(cbrt((double)nblockrows/128));
    dim3 grid_new(gridDim_new, gridDim_new, gridDim_new);

    int *runtime;
    int *load;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);

    hipMalloc(&load, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(load, nblockrows, 0);
#endif

    // ------
    GpuTimer bmv_timer;
    bmv_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

//        bmv8_sparse<int, float><<<grid, 32>>>(tA, tB, fC, bsrRowPtr, bsrColInd, nblockrows, nblocks, runtime, load);
        bmv8_sparse_sharedvector<int, float><<<grid_new, 1024>>>(tA, tB, fC, bsrRowPtr, bsrColInd, nblockrows, nblocks);
    }

    bmv_timer.Stop();
    double bmv8_time = bmv_timer.ElapsedMillis()/double(TEST_TIMES);
    // ------
#ifdef PROF
    printTimenLoadReport<<<1,1>>>(runtime, load, nblockrows); hipFree(runtime); hipFree(load);
#endif


    // free storage
    hipFree(tA);
    hipFree(tB);

    // copy result to host for verification
    float* result_bsrbmv8 = (float*)malloc(nrows * 1 * sizeof(float)); // don't care padding result
    hipMemcpy(result_bsrbmv8, fC, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("result_bsrbmv8: \n"); printResVec<float><<<1,1>>>(fC, nrows);
    printf("bsrbmv8 nnz in vec: %d\n", countNnzinVec<float>(result_bsrbmv8, nrows));
#endif

    // ============================================= cuSPARSE csr spmv-float
    // metadata for cuSPARSE API
    hipsparseHandle_t handle_csr;
    hipsparseMatDescr_t mat_A;
    hipsparseStatus_t cusparse_status;

    hipsparseCreate(&handle_csr);
    hipsparseCreateMatDescr(&mat_A);
    hipsparseSetMatType(mat_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(mat_A, HIPSPARSE_INDEX_BASE_ZERO);

    // dummy multiplication variables
    // y = α ∗ op ( A ) ∗ x + β ∗ y
#if TEST_TIMES > 1
    float alpha = 1.0, beta = 1.0;
#else
    float alpha = 1.0, beta = 0.0;
#endif

    // create dense vector storage
    float *dX, *dY;
    hipMalloc((void**)&dX, sizeof(float)*nrows);
    hipMemcpy(dX, B, sizeof(float)*nrows, hipMemcpyHostToDevice);  // [nrows] to [nb * blocksize] (paddings) is not moved
    hipMalloc((void**)&dY, sizeof(float)*nrows);
    setDeviceValArr<int, float><<<1,1>>>(dY, nrows, 0);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrmv(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, ncols, nnz,
                    &alpha, mat_A, csrVal, csrRowPtr, csrColInd, dX, &beta, dY);
    }

    csr_timer.Stop();
    double cusparsecsrspmvfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    // copy result to host for verification
    float* result_cusparsecsrspmvfloat = (float*)malloc(nrows * 1 * sizeof(float));
    hipMemcpy(result_cusparsecsrspmvfloat, dY, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("csrspmvvec: \n"); printResVec<float><<<1,1>>>(dY, nrows);
    printf("cuSPARSE nnz in vec: %d\n", countNnzinVec<float>(result_cusparsecsrspmvfloat, nrows));
#endif

    //============================================= check result
    // verify bsrbmv with cuSPARSE baseline
    printf("BSR BMV-8 success: %d\n", checkResult<float>(result_bsrbmv8, result_cusparsecsrspmvfloat, nrows));

    // print time
    printf("BSR BMV-8: %.3lf\n", bmv8_time);
    printf("CuSPARSE CSR SpMV-float: %.3lf\n", cusparsecsrspmvfloat_time);

    //============================================= free memory
    // free cusparse bsr metadata
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // free cusparse csr spmv
    hipsparseDestroyMatDescr(mat_A);
    hipsparseDestroy(handle_csr);
    hipFree(dX);
    hipFree(dY);

    // free mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);

    hipFree(fB);
    hipFree(fC);

    // free all results
    free(result_bsrbmv8);
    free(result_cusparsecsrspmvfloat);
}

/* bsrbmv-16 */
int main16(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 16;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 2 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    ushort* tA;
    hipMalloc((void**)&tA, ceil((float)nblocks/4) * 4 * blocksize * sizeof(ushort)); // <--

#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit16Col<float><<<dim3(1, ceil((float)temp_nblocks/4)), 32>>>(bsrVal, tA, nblocks); // sparse matrix
    //printGlobalBSRBlock16<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_16(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
//    printGlobalBSRBlock16<<<1,1>>>(tA, blocksize, 5);
#endif

    // ============================================= input vector storage
    // generate random vector
    srand(time(0));
	float *B = (float*)malloc((nblockrows * blocksize) * 1 * sizeof(float));
	for (int i = 0; i < (nblockrows * blocksize) * 1; i ++)
    {
        float x = (float)rand() / RAND_MAX;
        if (i >= ncols) B[i] = 0;
        else B[i] = (x > 0.5) ? 1 : 0;
    }

#ifdef VERBOSE
    printf("initialize a vector with size %d x 1\n", (nblockrows * blocksize));
//    printf("orivec: \n"); printHostVec(B, (nblockrows * blocksize));
#endif

    // copy to device
	float *fB;
	hipMalloc(&fB, (nblockrows * blocksize) * 1 * sizeof(float));
	hipMemcpy(fB, B, (nblockrows * blocksize) * 1 * sizeof(float), hipMemcpyHostToDevice);

    // pack B
    ushort *tB;
    hipMalloc(&tB, ceil((float)nblockrows/2)* 2 * sizeof(ushort));
    setDeviceValArr<int, ushort><<<1,1>>>(tB, ceil((float)nblockrows/2)*2, 0);

    // get gridDim, this is to avoid nblockrows being larger than MAX_gridDim (65535?!)
    int gridDim = (int)ceil(cbrt((double)nblockrows/2));
    dim3 grid(gridDim, gridDim, gridDim);

#ifdef VERBOSE
    printf("ceil(nblockrows/2) = %d, cbrt(nblockrows/2) = %d\n", (int)ceil((double)nblockrows/2), gridDim);
#endif

    ToBit16Row<float><<<grid, 32>>>(fB, tB, nblockrows); // dense vector
//    printf("binarized vec: \n"); printBin16Vec<<<1,1>>>(tB, nblockrows);


    // ============================================= BSTC-16 bsr bmv
    // init C (result storage)
    float *fC;
    hipMalloc(&fC, (nblockrows * blocksize) * 1 * sizeof(float));
    setDeviceValArr<int, float><<<1,1>>>(fC, nblockrows * blocksize, 0);

    int gridDim_new = (int)ceil(cbrt((double)nblockrows/64));
    dim3 grid_new(gridDim_new, gridDim_new, gridDim_new);

    int *runtime;
    int *load;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);

    hipMalloc(&load, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(load, nblockrows, 0);
#endif

    // ------
    GpuTimer bmv_timer;
    bmv_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        //bmv16_sparse<int, float><<<grid, 32>>>(tA, tB, fC, bsrRowPtr, bsrColInd, nblockrows, nblocks, runtime, load);
        bmv16_sparse_sharedvector<int, float><<<grid_new, 1024>>>(tA, tB, fC, bsrRowPtr, bsrColInd, nblockrows, nblocks);
    }

    bmv_timer.Stop();
    double bmv16_time = bmv_timer.ElapsedMillis()/double(TEST_TIMES);
    // ------
#ifdef PROF
    printTimenLoadReport<<<1,1>>>(runtime, load, nblockrows); hipFree(runtime); hipFree(load);
#endif


    // free storage
    hipFree(tA);
    hipFree(tB);

    // copy result to host for verification
    float* result_bsrbmv16 = (float*)malloc(nrows * 1 * sizeof(float)); // don't care padding result
    hipMemcpy(result_bsrbmv16, fC, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("result_bsrbmv16: \n"); printResVec<float><<<1,1>>>(fC, nrows);
    printf("bsrbmv16 nnz in vec: %d\n", countNnzinVec<float>(result_bsrbmv16, nrows));
#endif

    // ============================================= cuSPARSE csr spmv-float
    // metadata for cuSPARSE API
    hipsparseHandle_t handle_csr;
    hipsparseMatDescr_t mat_A;
    hipsparseStatus_t cusparse_status;

    hipsparseCreate(&handle_csr);
    hipsparseCreateMatDescr(&mat_A);
    hipsparseSetMatType(mat_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(mat_A, HIPSPARSE_INDEX_BASE_ZERO);

    // dummy multiplication variables
    // y = α ∗ op ( A ) ∗ x + β ∗ y
#if TEST_TIMES > 1
    float alpha = 1.0, beta = 1.0;
#else
    float alpha = 1.0, beta = 0.0;
#endif

    // create dense vector storage
    float *dX, *dY;
    hipMalloc((void**)&dX, sizeof(float)*nrows);
    hipMemcpy(dX, B, sizeof(float)*nrows, hipMemcpyHostToDevice);  // [nrows] to [nb * blocksize] (paddings) is not moved
    hipMalloc((void**)&dY, sizeof(float)*nrows);
    setDeviceValArr<int, float><<<1,1>>>(dY, nrows, 0);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrmv(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, ncols, nnz,
                    &alpha, mat_A, csrVal, csrRowPtr, csrColInd, dX, &beta, dY);
    }

    csr_timer.Stop();
    double cusparsecsrspmvfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    // copy result to host for verification
    float* result_cusparsecsrspmvfloat = (float*)malloc(nrows * 1 * sizeof(float));
    hipMemcpy(result_cusparsecsrspmvfloat, dY, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("csrspmvvec: \n"); printResVec<float><<<1,1>>>(dY, nrows);
    printf("cuSPARSE nnz in vec: %d\n", countNnzinVec<float>(result_cusparsecsrspmvfloat, nrows));
#endif

    //============================================= check result
    // verify bsrbmv with cuSPARSE baseline
    printf("BSR BMV-16 success: %d\n", checkResult<float>(result_bsrbmv16, result_cusparsecsrspmvfloat, nrows));

    // print time
    printf("BSR BMV-16: %.3lf\n", bmv16_time);
    printf("CuSPARSE CSR SpMV-float: %.3lf\n", cusparsecsrspmvfloat_time);

    //============================================= free memory
    // free cusparse bsr metadata
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // free cusparse csr spmv
    hipsparseDestroyMatDescr(mat_A);
    hipsparseDestroy(handle_csr);
    hipFree(dX);
    hipFree(dY);

    // free mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);

    hipFree(fB);
    hipFree(fC);

    // free all results
    free(result_bsrbmv16);
    free(result_cusparsecsrspmvfloat);
}

/* bsrbmv-32 */
int main32(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 4 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));

#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit32Col<float><<<dim3(1, nblocks), 32>>>(bsrVal, tA, blocksize, nblocks * blocksize); // sparse matrix
//    printGlobalBSRBlock32<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_32(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
#endif

    // ============================================= input vector storage
    // generate random vector
    srand(time(0));
	float *B = (float*)malloc((nblockrows * blocksize) * 1 * sizeof(float));
	for (int i = 0; i < (nblockrows * blocksize) * 1; i ++)
    {
        float x = (float)rand() / RAND_MAX;
        if (i >= ncols) B[i] = 0;
        else B[i] = (x > 0.5) ? 1 : 0;
    }

#ifdef VERBOSE
    printf("initialize a vector with size %d x 1\n", (nblockrows * blocksize));
//    printf("orivec: \n"); printHostVec(B, (nblockrows * blocksize));
#endif

    // copy to device
	float *fB;
	hipMalloc(&fB, (nblockrows * blocksize) * 1 * sizeof(float));
	hipMemcpy(fB, B, (nblockrows * blocksize) * 1 * sizeof(float), hipMemcpyHostToDevice);

    // pack B
    unsigned *tB;
    hipMalloc(&tB, nblockrows * 1 * sizeof(unsigned)); // (nblockrows * blocksize) / 32 = nblockrows

    // get gridDim, this is to avoid nblockrows being larger than MAX_gridDim (65535?!)
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);

#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

    ToBit32Row<float><<<grid, 32>>>(fB, tB, nblockrows * blocksize, 1, nblockrows); // dense vector


    // ============================================= BSTC-32 bsr bmv
    // init C (result storage)
    float *fC;
    hipMalloc(&fC, (nblockrows * blocksize) * 1 * sizeof(float));
    setDeviceValArr<int, float><<<1,1>>>(fC, nblockrows * blocksize, 0);

    int *runtime;
    int *load;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);

    hipMalloc(&load, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(load, nblockrows, 0);
#endif

    // ------
    GpuTimer bmv_timer;
    bmv_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmv32_sparse<int, float><<<grid, 32>>>(tA, tB, fC, blocksize, nblocks, 1, bsrRowPtr, bsrColInd, nblockrows, nblocks, runtime, load);
    }

    bmv_timer.Stop();
    double bmv32_time = bmv_timer.ElapsedMillis()/double(TEST_TIMES);
    // ------
#ifdef PROF
    printTimenLoadReport<<<1,1>>>(runtime, load, nblockrows); hipFree(runtime); hipFree(load);
#endif


    // free storage
    hipFree(tA);
    hipFree(tB);

    // copy result to host for verification
    float* result_bsrbmv32 = (float*)malloc(nrows * 1 * sizeof(float)); // don't care padding result
    hipMemcpy(result_bsrbmv32, fC, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("result_bsrbmv32: \n"); printResVec<float><<<1,1>>>(fC, nrows);
    printf("bsrbmv32 nnz in vec: %d\n", countNnzinVec<float>(result_bsrbmv32, nrows));
#endif

    // ============================================= cuSPARSE csr spmv-float
    // metadata for cuSPARSE API
    hipsparseHandle_t handle_csr;
    hipsparseMatDescr_t mat_A;
    hipsparseStatus_t cusparse_status;

    hipsparseCreate(&handle_csr);
    hipsparseCreateMatDescr(&mat_A);
    hipsparseSetMatType(mat_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(mat_A, HIPSPARSE_INDEX_BASE_ZERO);

    // dummy multiplication variables
    // y = α ∗ op ( A ) ∗ x + β ∗ y
#if TEST_TIMES > 1
    float alpha = 1.0, beta = 1.0;
#else
    float alpha = 1.0, beta = 0.0;
#endif

    // create dense vector storage
    float *dX, *dY;
    hipMalloc((void**)&dX, sizeof(float)*nrows);
    hipMemcpy(dX, B, sizeof(float)*nrows, hipMemcpyHostToDevice);  // [nrows] to [nb * blocksize] (paddings) is not moved
    hipMalloc((void**)&dY, sizeof(float)*nrows);
    setDeviceValArr<int, float><<<1,1>>>(dY, nrows, 0);

    // ------

    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrmv(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, ncols, nnz,
                    &alpha, mat_A, csrVal, csrRowPtr, csrColInd, dX, &beta, dY);
    }

    csr_timer.Stop();
    double cusparsecsrspmvfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    // copy result to host for verification
    float* result_cusparsecsrspmvfloat = (float*)malloc(nrows * 1 * sizeof(float));
    hipMemcpy(result_cusparsecsrspmvfloat, dY, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("csrspmvvec: \n"); printResVec<float><<<1,1>>>(dY, nrows);
    printf("cuSPARSE nnz in vec: %d\n", countNnzinVec<float>(result_cusparsecsrspmvfloat, nrows));
#endif

    //============================================= check result
    // verify bsrbmv with cuSPARSE baseline
    printf("BSR BMV-32 success: %d\n", checkResult<float>(result_bsrbmv32, result_cusparsecsrspmvfloat, nrows));

    // print time
    printf("BSR BMV-32: %.3lf\n", bmv32_time);
    printf("CuSPARSE CSR SpMV-float: %.3lf\n", cusparsecsrspmvfloat_time);

    //============================================= free memory
    // free cusparse bsr metadata
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // free cusparse csr spmv
    hipsparseDestroyMatDescr(mat_A);
    hipsparseDestroy(handle_csr);
    hipFree(dX);
    hipFree(dY);

    // free mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);

    hipFree(fB);
    hipFree(fC);

    // free all results
    free(result_bsrbmv32);
    free(result_cusparsecsrspmvfloat);
}

/* bsrbmv-64 */
int main64(int argc, char* argv[])
{

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 64;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate nblocks first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 8 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // packed matrix tA
    ullong* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(ullong));


#ifdef NONBATCH
    // for small matrices: csr2bsr directly
    hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
    hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
                csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);

    // pack A
    ToBit64Col<float><<<dim3(1, nblocks), 32>>>(bsrVal, tA, blocksize, nblocks * blocksize); // sparse matrix
//    printGlobalBSRBlock64<<<1,1>>>(tA, blocksize, nblocks);

    // free memory
    hipFree(bsrVal);
#else
    // use batch transform as default
    csr2bsr_batch_64(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                     bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);
#endif

    // ============================================= input vector storage
    // generate random vector
    srand(time(0));
	float *B = (float*)malloc((nblockrows * blocksize) * 1 * sizeof(float));
	for (int i = 0; i < (nblockrows * blocksize) * 1; i ++)
    {
        float x = (float)rand() / RAND_MAX;
        if (i >= ncols) B[i] = 0;
        else B[i] = (x > 0.5) ? 1 : 0;
    }

#ifdef VERBOSE
    printf("initialize a vector with size %d x 1\n", (nblockrows * blocksize));
//    printf("orivec: \n"); printHostVec(B, (nblockrows * blocksize));
#endif

    // copy to device
	float *fB;
	hipMalloc(&fB, (nblockrows * blocksize) * 1 * sizeof(float));
	hipMemcpy(fB, B, (nblockrows * blocksize) * 1 * sizeof(float), hipMemcpyHostToDevice);

    // pack B
    ullong *tB;
    hipMalloc(&tB, nblockrows * 1 * sizeof(ullong)); // (nblockrows * blocksize) / 64 = nblockrows

    // get gridDim, this is to avoid nblockrows being larger than MAX_gridDim (65535?!)
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);

#ifdef VERBOSE
    printf("cbrt(nblockrows) = %d\n", gridDim);
#endif

    ToBit64Row<float><<<grid, 32>>>(fB, tB, nblockrows * blocksize, 1, nblockrows); // dense vector


    // ============================================= BSTC-64 bsr bmv
    // init C (output storage)
    float *fC;
    hipMalloc(&fC, (nblockrows * blocksize) * 1 * sizeof(float));
    setDeviceValArr<int, float><<<1,1>>>(fC, nblockrows * blocksize, 0);

    int *runtime;
#ifdef PROF
    hipMalloc(&runtime, nblockrows * sizeof(int));
    setDeviceValArr<int, int><<<1,1>>>(runtime, nblockrows, 0);
#endif

    // ------

    GpuTimer bmv_timer;
    bmv_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmv64_sparse<int, float><<<grid, 32>>>(tA, tB, fC, blocksize, nblocks, 1, bsrRowPtr, bsrColInd, nblockrows, nblocks, runtime);
    }

    bmv_timer.Stop();
    double bmv64_time = bmv_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

#ifdef PROF
    printTimeReport<<<1,1>>>(runtime, nblockrows); hipFree(runtime);
#endif

    // free memory
    hipFree(tA);
    hipFree(tB);

    // copy result to host for verification
    float* result_bsrbmv64 = (float*)malloc(nrows * 1 * sizeof(float)); // don't care padding result
    hipMemcpy(result_bsrbmv64, fC, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("result_bsrbmv32: \n"); printResVec<float><<<1,1>>>(fC, nrows);
    printf("bsrbmv64 nnz in vec: %d\n", countNnzinVec<float>(result_bsrbmv64, nrows));
#endif

    // ============================================= cuSPARSE csr spmv-float
    // metadata for cuSPARSE API
    hipsparseHandle_t handle_csr;
    hipsparseMatDescr_t mat_A;
    hipsparseStatus_t cusparse_status;

    hipsparseCreate(&handle_csr);
    hipsparseCreateMatDescr(&mat_A);
    hipsparseSetMatType(mat_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(mat_A, HIPSPARSE_INDEX_BASE_ZERO);

    // dummy multiplication variables
    // y = α ∗ op ( A ) ∗ x + β ∗ y
#if TEST_TIMES > 1
    float alpha = 1.0, beta = 1.0;
#else
    float alpha = 1.0, beta = 0.0;
#endif

    // create dense vector
    float *dX, *dY;
    hipMalloc((void**)&dX, sizeof(float)*nrows);
    hipMemcpy(dX, B, sizeof(float)*nrows, hipMemcpyHostToDevice);  // [nrows] to [nb * blocksize] (paddings) is not moved
    hipMalloc((void**)&dY, sizeof(float)*nrows);
    setDeviceValArr<int, float><<<1,1>>>(dY, nrows, 0);

    // ------
    GpuTimer csr_timer;
    csr_timer.Start();

    for (int i=0; i<TEST_TIMES; i++) {
        hipsparseScsrmv(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, ncols, nnz,
                    &alpha, mat_A, csrVal, csrRowPtr, csrColInd, dX, &beta, dY);
    }

    csr_timer.Stop();
    double cusparsecsrspmvfloat_time = csr_timer.ElapsedMillis()/double(TEST_TIMES);

    // ------

    // copy result to host for verification
    float* result_cusparsecsrspmvfloat = (float*)malloc(nrows * 1 * sizeof(float));
    hipMemcpy(result_cusparsecsrspmvfloat, dY, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);

#ifdef VERBOSE
//    printf("csrspmvvec: \n"); printResVec<float><<<1,1>>>(dY, nrows);
    printf("cuSPARSE nnz in vec: %d\n", countNnzinVec<float>(result_cusparsecsrspmvfloat, nrows));
#endif

    //============================================= check result
    printf("BSR BMV-64 success: %d\n", checkResult<float>(result_bsrbmv64, result_cusparsecsrspmvfloat, nrows));

    printf("BSR BMV-64: %.3lf\n", bmv64_time);
    printf("CuSPARSE CSR SpMV-float: %.3lf\n", cusparsecsrspmvfloat_time);


    //============================================= free memory
    // free descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // free cusparse csr spmv
    hipsparseDestroyMatDescr(mat_A);
    hipsparseDestroy(handle_csr);
    hipFree(dX);
    hipFree(dY);

    // free mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);

    hipFree(fB);
    hipFree(fC);

    // free all results
    free(result_bsrbmv64);
    free(result_cusparsecsrspmvfloat);

}

int main(int argc, char* argv[])
{
#if BLOCKSIZE == 64
    main64(argc, argv);
#elif BLOCKSIZE == 32
    main32(argc, argv);
#elif BLOCKSIZE == 16
    main16(argc, argv);
#else
    main8(argc, argv);
#endif
}
