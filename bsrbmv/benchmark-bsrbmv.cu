#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define TEST_TIMES 5 // do not test more than 1 time for now (vector not clean)
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

//#include "bsrbmv.cu"
#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"

bool check_result(float* p1, float* p2, const int N)
{
    bool flag = true;
    for (int i = 0; i < N; i ++) {

        float diff = p1[i] - p2[i];
        if (fabs(diff) > 1e-6) {
            printf("[%d](%.f,%.f),", i, p1[i], p2[i]);
            flag = false;
        }
    }
    return flag;
}

bool check_result(float* p1, int* p2, const int N)
{
    bool flag = true;
    for (int i = 0; i < N * N; i ++) {
        //printf("(%.0f,%d)",p1[i],p2[i]);
        float diff = p1[i] - (float)p2[i];
        if (fabs(diff) > 1e-6) {
            flag = false;
        }
    }
    return flag;
}

int countnnzinvec(const float* vec, const int N)
{
    int counter = 0;
    for (int i=0; i<N; i++) if (vec[i] != 0) counter += 1;
    return counter;
}

void printvec(float* vec, const int N)
{
    for(int i=0; i<N; i++) printf(vec[i]>0?"1":"0");
    printf("\n");
}

void printresvec(float* vec, const int N)
{
    for(int i=0; i<N; i++) printf("%d", (int)vec[i]);
    printf("\n");
}

void printmat(float* bsrval, const int nblocks, const int blocksize)
{
    for(int i=0; i<nblocks; i++) {
        printf("[%d]\n", i);
        for(int j=0; j<blocksize; j++) {
            for(int k=0; k<blocksize; k++) {
                printf(bsrval[i*blocksize*blocksize+j*blocksize+k]>0?"1":"0");
            }
            printf("\n");
        }
        printf("\n");
    }
}

void printbinvec(unsigned* binvec, const int N)
{
    for(int i=0; i<N; i++) bin(binvec[i]);
    printf("\n");
}

void printbinmat(unsigned* binbsrval, const int nblocks, const int blocksize)
{
    for(int i=0; i<nblocks; i++) {
        printf("[%d]\n", i);
        for(int j=0; j<blocksize; j++) {
            bin(binbsrval[i*blocksize+j]);
            printf("\n");
        }
        printf("\n");
    }
}

void printind(int* indarr, const int N)
{
    for(int i=0; i<N; i++) {
        printf("%d ", indarr[i]);
    }
    printf("\n");
}

int main(int argc, char* argv[])
{

//    bool trans_A = false;
//    bool trans_B = false;

    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // matrix storage -----------------------------------
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_COLUMN;

    // csr2bsr in column-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);

    // packed matrix
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));

//    if (nblocks > 100000) { // Large Matrices: batch csr2bsr & pack A at the same time
        csr2bsr_batch_32(h_csrRowPtr, h_csrColInd, nrows, ncols, nnz,
                      bsrRowPtr, bsrColInd, tA, blocksize, nblockrows, nblocks);

//    } else { // Small Matrices: csr2bsr & pack A
//       hipMalloc((void**)&bsrVal, sizeof(float)*(blocksize*blocksize)*nblocks);
//       hipsparseScsr2bsr(handle, dirA, nrows, ncols, csr_descr, csrVal,
//                    csrRowPtr, csrColInd, blocksize, bsr_descr, bsrVal, bsrRowPtr, bsrColInd);
//
//      // pack A
//      ToBit32Col<float><<<dim3(CEIL(blocksize), CEIL(nblocks * blocksize)), 32>>>(bsrVal, tA, blocksize, nblocks * blocksize); // sparse matrix
//
//    }

    // input vector and result vector storage -----------------------------------
    // generate random vector
    srand(time(0));
	float *B = (float*)malloc((nblockrows * blocksize) * 1 * sizeof(float));
	for (int i = 0; i < (nblockrows * blocksize) * 1; i ++)
    {
        float x = (float)rand() / RAND_MAX;
        if (i >= ncols) B[i] = 0;
        else B[i] = (x > 0.5) ? 1 : 0;
    }
    printf("initialize a vector with size %d x 1\n", (nblockrows * blocksize));
//    printf("orivec: \n"); printvec(B, (nblockrows * blocksize));

    // copy to device
	float *fB;
	hipMalloc(&fB, (nblockrows * blocksize) * 1 * sizeof(float));
	hipMemcpy(fB, B, (nblockrows * blocksize) * 1 * sizeof(float), hipMemcpyHostToDevice);

    // pack B
    unsigned *tB;
    hipMalloc(&tB, nblockrows * 1 * sizeof(unsigned)); // (nblockrows * blocksize) / 32 = nblockrows
    ToBit32Row<float><<<dim3(CEIL(nblockrows * blocksize), CEIL(1)), 32>>>(fB, tB, nblockrows * blocksize, 1); // dense vector

	// time measurement setup -----------------------------------
	hipEvent_t start, stop;
	float milliseconds = 0;

    // ============================================= BSTC-32 bsr bmv
    // init C
    float *fC;
    hipMalloc(&fC, (nblockrows * blocksize) * 1 * sizeof(float));
    hipMemset(fC, 0, (nblockrows * blocksize) * 1 * sizeof(float));

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // get grid dim
    double nbr = cbrt((double)nblockrows);
    int blockdim = (int)ceil(nbr);
    printf("cbrt(nblockrows) = %d\n", blockdim);
    dim3 grid(blockdim, blockdim, blockdim);

    // ------
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++) { // follow warp consolidation model (32 threads per block)

        bmv32_sparse<int, float><<<grid, 32>>>(tA, tB, fC, blocksize, nblocks, 1, bsrRowPtr, bsrColInd, nblockrows, nblocks);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmv32_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    // ------

    float* result_bsrbmv32 = (float*)malloc(nrows * 1 * sizeof(float)); // don't care padding result
    hipMemcpy(result_bsrbmv32, fC, nrows * 1 * sizeof(float), hipMemcpyDeviceToHost);
    //printf("result_bsrbmv32: \n"); printvec(result_bsrbmv32, ncols);
    printf("nnz in vec: %d\n", countnnzinvec(result_bsrbmv32, nrows));

    // ============================================= cuSPARSE bsr spmv-float 32
    // y = α ∗ op ( A ) ∗ x + β ∗ y
//    // allocate vector x and vector y large enough for bsrmv
//    float *x, *y;
//    hipMalloc((void**)&x, sizeof(float)*(nblockrows * blocksize));
//    hipMemcpy(x, fB, sizeof(float)*ncols, hipMemcpyHostToDevice);  // [ncols] to [nb * blocksize] (paddings) is not moved
//    hipMalloc((void**)&y, sizeof(float)*(nblockrows * blocksize));
//    hipMemset(y, 0, sizeof(float)*ncols);
//
//    // perform bsrmv
//    float alpha = 1.0, beta = 0.0;
//    hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
//
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//    // ------
//    hipEventRecord(start);
//    for (int i=0; i<TEST_TIMES; i++) {
//        hipsparseSbsrmv(handle, dirA, transA, mb, nb, nblocks, &alpha,
//                    bsr_descr, bsrVal, bsrRowPtr, bsrColInd, blocksize, x, &beta, y);
//    }
//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//
//    milliseconds = 0;
//    hipEventElapsedTime(&milliseconds,start,stop);
//    double cusparsebsrspmvfloat_time = (milliseconds*1e3)/double(TEST_TIMES);
//    // ------
//
//    float* result_cusparsebsrspmvfloat = (float*)malloc(ncols * 1 * sizeof(float));
//    hipMemcpy(result_cusparsebsrspmvfloat, y, ncols * 1 * sizeof(float), hipMemcpyDeviceToHost);
//    //printf("baselinevec: \n"); printresvec(result_cusparsebsrspmvfloat, ncols);

    // ============================================= cuSPARSE csr spmv-float
//    hipsparseHandle_t handle_csr;
//    hipsparseMatDescr_t mat_A;
//    hipsparseStatus_t cusparse_status;
//
//    hipsparseCreate(&handle_csr);
//    hipsparseCreateMatDescr(&mat_A);
//    hipsparseSetMatType(mat_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
//    hipsparseSetMatIndexBase(mat_A, HIPSPARSE_INDEX_BASE_ZERO);
//
//    // create dense vector
//    float *dX, *dY;
//    hipMalloc((void**)&dX, sizeof(float)*(nblockrows * blocksize));
//    hipMemcpy(dX, fB, sizeof(float)*ncols, hipMemcpyHostToDevice);  // [ncols] to [nb * blocksize] (paddings) is not moved
//    hipMalloc((void**)&dY, sizeof(float)*(nblockrows * blocksize));
//    hipMemset(dY, 0, sizeof(float)*ncols);
//
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
//
//    // ------
//    hipEventRecord(start);
//    for (int i=0; i<TEST_TIMES; i++) {
//        hipsparseScsrmv(handle_csr, HIPSPARSE_OPERATION_NON_TRANSPOSE, nrows, ncols, nnz,
//                    &alpha, mat_A, csrVal, csrRowPtr, csrColInd, dX, &beta, dY);
//    }
//
//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//
//    milliseconds = 0;
//    hipEventElapsedTime(&milliseconds,start,stop);
//    double cusparsecsrspmvfloat_time = (milliseconds*1e3)/double(TEST_TIMES);
//    // ------
//
//    float* result_cusparsecsrspmvfloat = (float*)malloc(ncols * 1 * sizeof(float));
//    hipMemcpy(result_cusparsecsrspmvfloat, dY, ncols * 1 * sizeof(float), hipMemcpyDeviceToHost);
//    //printf("csrspmvvec: \n"); printresvec(result_cusparsecsrspmvfloat, ncols);


    //============================================= CHECK RESULT
    //printf("CuSPARSE BSR SpMV-float (baseline) success: %d\n", check_result(result_cusparsebsrspmvfloat, result_cusparsebsrspmvfloat, ncols));
    //printf("BSR BMV-32 success: %d\n", check_result(result_bsrbmv32, result_cusparsebsrspmvfloat, ncols));

    printf("BSR BMV-32: %.3lf\n", bmv32_time);
//    printf("CuSPARSE BSR SpMV-float: %.3lf\n", cusparsebsrspmvfloat_time);
//    printf("CuSPARSE CSR SpMV-float: %.3lf\n", cusparsecsrspmvfloat_time);

    // free descr and handle memory
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // free cusparse bsr spmv
//    hipFree(x);
//    hipFree(y);
//
//    // free cusparse csr spmv
//    hipsparseDestroyMatDescr(mat_A);
//    hipsparseDestroy(handle_csr);
//    hipFree(dX);
//    hipFree(dY);

    // free mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);

    hipFree(fB);
    hipFree(fC);

    // free all results
    free(result_bsrbmv32);
//    free(result_cusparsebsrspmvfloat);
//    free(result_cusparsecsrspmvfloat);

}
