#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define MAX_ITER 10
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#include "mmio_highlevel.h"
#include "csr2bsr_batch.cu"

/* bsrbmv-32 */
int main32(int argc, char* argv[])
{
    hipSetDevice(0);
    if (argc < 2)
    {
        printf("./exe [xxx.mtx]\n");
        exit(1);
    }

    // ============================================= matrix storage
    // read sparse matrix from file and store as csr format
    // matrix metadata
    char *filename = argv[1]; // e.g. "G43.mtx"
    printf("input sparse matrix: %s\n", filename);

    int nrows, ncols, nnz, isSymmetric;
    mmio_info<float>(&nrows, &ncols, &nnz, &isSymmetric, filename);
    printf("nrows: %d, ncols: %d, nnz: %d, isSymmetric: ", nrows, ncols, nnz); printf(isSymmetric?"true\n":"false\n");
    unsigned csrbytes = (nrows+1+nnz*2) * 4;
    printf("csr total size: "); printBytes(csrbytes); printf("\n");

    // matrix csr in host
    int* h_csrRowPtr, *h_csrColInd;
    float* h_csrVal;
    h_csrRowPtr = (int*) malloc(sizeof(int) * (nrows+1));
    h_csrColInd = (int*) malloc(sizeof(int) * nnz);
    h_csrVal = (float*) malloc(sizeof(float) * nnz);
    mmio_data<float>(h_csrRowPtr, h_csrColInd, h_csrVal, filename);

    // copy csr to device
    int* csrRowPtr, *csrColInd;
    float* csrVal;
    hipMalloc(&csrRowPtr, sizeof(int) * (nrows+1));
    hipMalloc(&csrColInd, sizeof(int) * nnz);
    hipMalloc(&csrVal, sizeof(float) * nnz);
    hipMemcpy(csrRowPtr, h_csrRowPtr, sizeof(int) * (nrows+1), hipMemcpyHostToDevice);
    hipMemcpy(csrColInd, h_csrColInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(csrVal, h_csrVal, sizeof(float) * nnz, hipMemcpyHostToDevice);
    // force all csrval to be 1 (this is for handling weighted adjacency matrix)
    setDeviceValArr<int, float><<<1,1>>>(csrVal, nnz, 1.0);

	// transform from csr to bsr using cuSPARSE
	int* bsrRowPtr, *bsrColInd;
	float* bsrVal;
	int blocksize = 32;
	int mb = (nrows + blocksize-1)/blocksize;
    int nb = (ncols + blocksize-1)/blocksize;
    int nblockrows = mb;

	// cuSPARSE API metadata setup
    hipsparseMatDescr_t csr_descr = 0;
    hipsparseCreateMatDescr(&csr_descr);
    hipsparseSetMatType(csr_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr_descr,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseMatDescr_t bsr_descr = 0;
    hipsparseCreateMatDescr(&bsr_descr);
    hipsparseSetMatType(bsr_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr_descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipStream_t streamId = 0;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, streamId);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;

    // csr2bsr in row-major order, estimate first
    int nblocks;

    hipMalloc((void**)&bsrRowPtr, sizeof(int) *(nblockrows+1));
    hipsparseXcsr2bsrNnz(handle, dirA, nrows, ncols, csr_descr,
                        csrRowPtr, csrColInd, blocksize, bsr_descr, bsrRowPtr, &nblocks);
    hipMalloc((void**)&bsrColInd, sizeof(int)*nblocks);
    printf("blocksize: %d, nblockrows: %d, nblocks: %d\n", blocksize, nblockrows, nblocks);
    unsigned bytes = (nblocks * blocksize * 4 + (nblockrows+1+nblocks) * 4);
    printf("bsr total size: "); printBytes(bytes); printf("\n");

    // csr2csc for B as A^T
    int* cscRowInd, *cscColPtr;
    float* cscVal;
    hipMalloc(&cscRowInd, sizeof(int) * nnz);
    hipMalloc(&cscColPtr, sizeof(int) * (nrows+1));
    hipMalloc(&cscVal, sizeof(float) * nnz);

    hipsparseHandle_t handle_csr2csc;
    hipsparseCreate(&handle_csr2csc);
    hipsparseScsr2csc(handle_csr2csc, nrows, ncols, nnz,
                     csrVal, csrRowPtr, csrColInd,
                     cscVal, cscRowInd, cscColPtr,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDestroy(handle_csr2csc);

    int *h_cscRowInd, *h_cscColPtr;
    h_cscRowInd = (int*) malloc(sizeof(int) * nnz);
    h_cscColPtr = (int*) malloc(sizeof(int) * (nrows+1));
    hipMemcpy(h_cscRowInd, cscRowInd, sizeof(int) * nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_cscColPtr, cscColPtr, sizeof(int) * (nrows+1), hipMemcpyDeviceToHost);

    // csr2bsr for B & pack matrix for tB
    int* new_bsrRowPtr, *new_bsrColInd;
    hipMalloc(&new_bsrRowPtr, sizeof(int) * (nblockrows+1));
    hipMalloc(&new_bsrColInd, sizeof(int) * nblocks);

    // packed matrix
    unsigned* tA;
    hipMalloc((void**)&tA, nblocks * blocksize * sizeof(unsigned));

    // use batch transform as default
    csr2bsr_batch_32(h_cscColPtr, h_cscRowInd, nrows, ncols, nnz,
                     new_bsrRowPtr, new_bsrColInd, tA, blocksize, nblockrows, nblocks);

    free(h_cscRowInd);
    free(h_cscColPtr);

    // ============================================= input vector storage

    // get gridDim, this is to avoid nblockrows being larger than MAX_gridDim (65535?!)
    int gridDim = (int)ceil(cbrt((double)nblockrows));
    dim3 grid(gridDim, gridDim, gridDim);

    // ============================================= BSTC-32 bsr bmv
    // difference vector
    float* diff;
    hipMalloc((void**)&diff, nblockrows * blocksize * sizeof(float));
    setDeviceValArr<int, float><<<1,1>>>(diff, nblockrows * blocksize , 0);

    // parent vector
    float* parent;
    hipMalloc((void**)&parent, nblockrows * blocksize * sizeof(float));
    float* parent_temp;
    hipMalloc((void**)&parent_temp, nblockrows * blocksize * sizeof(float));

    // grand parent vector
    float* grandparent;
    hipMalloc((void**)&grandparent, nblockrows * blocksize * sizeof(float));
    float* grandparent_temp;
    hipMalloc((void**)&grandparent_temp, nblockrows * blocksize * sizeof(float));

    // Min neighbor grandparent vector
    float* min_neighbor_parent;
    hipMalloc((void**)&min_neighbor_parent, nblockrows * blocksize * sizeof(float));
    float* min_neighbor_parent_temp;
    hipMalloc((void**)&min_neighbor_parent_temp, nblockrows * blocksize * sizeof(float));

    // Initialize parent and min_neighbor_parent to:
    // [0]:0 [1]:1 [2]:2 [3]:3 [4]:4, etc.
    // parent.fillAscending(A_nrows)
    // min_neighbor_parent.dup(&parent)
    // min_neighbor_parent_temp.dup(&parent)
    // grandparent.dup(&parent)
    // grandparent_temp.dup(&parent)

    int gridDim_new = (int)ceil(cbrt((double)nblockrows/32));
    dim3 grid_new(gridDim_new, gridDim_new, gridDim_new);

    printf("nrows: %d\n", nrows);
    printf("------------------------------------\n");

    int iter;
    int succ = 0;
    int *succptr;
    hipMalloc((void**)&succptr, sizeof(int));

     dim3 NT, NB;
     int nt = 1024;
     NT.x = nt;
     NT.y = 1;
     NT.z = 1;
     NB.x = (nblockrows+nt-1)/nt;
     NB.y = 1;
     NB.z = 1;

    // ------
    GpuTimer bmvbin_timer;
    double bmvbin32_time;
    bmvbin_timer.Start();

    for (iter=1; iter<=MAX_ITER; iter++) {

        // duplicate parent as parent_temp
        hipMemcpy(parent_temp, parent, nblockrows * blocksize * sizeof(float), hipMemcpyDeviceToDevice);

        // 1) Stochastic hooking.
        // mngf[u] = A x gf
        // mxv: minselectsecondsemiring <-- think of it as just mul for now

        // ---
        // solution 1
        // bmvbin_timer.Start();
        // bmv32_sparse_bin_masked_v1<int, float><<<NB, NT>>>(tA, frontier1, frontier2, new_bsrRowPtr, new_bsrColInd, nblockrows, visited);
        // bmvbin_timer.Stop();
        // bmvbin32_time += bmvbin_timer.ElapsedMillis();

        // solution 2
        // bmvbin_timer.Start();
        // bmv32_sparse_bin_masked_v2<int, float><<<grid, 32>>>(tA, frontier1, frontier2, new_bsrRowPtr, new_bsrColInd, nblockrows, visited);
        // // Mask<<<1,1>>>(frontier2, nblockrows, visited); <-- required only when masked is not pass in
        // bmvbin_timer.Stop();
        // bmvbin32_time += bmvbin_timer.ElapsedMillis();


        // solution 3
        // bmvbin_timer.Start();
        // bmv32_sparse_bin_masked_v3<int, float><<<grid, 32>>>(tA, frontier1, frontier2, new_bsrRowPtr, new_bsrColInd, nblockrows, visited);
        // bmvbin_timer.Stop();
        // bmvbin32_time += bmvbin_timer.ElapsedMillis();

        // solution 4
        // bmvbin_timer.Start();
        bmv32_sparse_full<int, float><<<grid_new, 1024>>>(tA, grandparent, min_neighbor_parent_temp, new_bsrRowPtr, new_bsrColInd, nblockrows);
        // bmvbin_timer.Stop();
        // bmvbin32_time += bmvbin_timer.ElapsedMillis();

        // solution 5
        // bmvbin_timer.Start();
        // bmv32_sparse_bin_masked_v5<int, float><<<grid_new, 1024>>>(tA, frontier1, frontier2, new_bsrRowPtr, new_bsrColInd, nblockrows, visited);
        // bmvbin_timer.Stop();
        // bmvbin32_time += bmvbin_timer.ElapsedMillis();

        // ---

        // ewiseadd: MinimumSelectSecondSemiring <-- and then see if mnp_temp is smaller than parent
        ewiseMin<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(min_neighbor_parent, nblockrows*blocksize,
                                                                   min_neighbor_parent, min_neighbor_parent_temp);


        // f[f[u]] = mngf[u]
        assignScatter<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(parent, nblockrows*blocksize,
                                                                        min_neighbor_parent, parent_temp);

        // 2) Aggressive hooking.
        // f = min(f, mngf)
        ewiseMin<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(parent, nblockrows*blocksize,
                                                                   parent, parent_temp);

        // 3) Shortcutting.
        // f = min(f, gf)
        ewiseMin<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(parent, nblockrows*blocksize,
                                                                   parent, min_neighbor_parent);

        // 4) Calculate grandparents.
        // gf[u] = f[f[u]]
        extractGather<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(grandparent, nblockrows*blocksize,
                                                                        parent, parent);

        // 5) Check termination.
        // eWiseMult, MinimumNotEqualToSemiring
        ewiseNotEqual<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(diff, nblockrows*blocksize, grandparent_temp, grandparent);

        // reduce add diff to succ
        resetSuccptr<<<1,1>>>(succptr); //<-- use together with reduce
        reduceAdd<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(succptr, nblockrows*blocksize, diff);
        hipMemcpy(&succ, succptr, sizeof(int), hipMemcpyDeviceToHost);

        // terminate condition
       if (succ == 0) break;

       // grandparent_temp.dup(&grandparent)
       hipMemcpy(grandparent_temp, grandparent, nblockrows * blocksize * sizeof(float), hipMemcpyDeviceToDevice);

       // 6) Similar to BFS and SSSP, we should filter out the unproductive
        // vertices from the next iteration.
        //     assign<int, bool, int, int>(&grandparent, &diff, GrB_NULL,
        // std::numeric_limits<int>::max(), GrB_ALL, A_nrows, desc); ,with mask
        assignMax<<<(int)ceil(nblockrows*blocksize/1024.0), 1024>>>(grandparent, nblockrows*blocksize, diff, std::numeric_limits<float>::max());

        printf("succ: %d\n", succ); // <-- print will slow down some time

       // copy result to output
       // v->dup(&parent)

    }

    bmvbin_timer.Stop();
    bmvbin32_time = bmvbin_timer.ElapsedMillis();

    printf("------------------------------------\n");
    int niter = iter; printf("niter: %d\n", niter);
    // ------

    // free storage
    hipFree(tA);

#ifdef VERBOSE
//    printf("result_bsrbmv32: \n"); printResVec<float><<<1,1>>>(fC, nrows);
//    printf("result_bsrbmv32-bin: \n"); printBin32Vec<<<1,1>>>(tC, nblockrows);
//    verify32BinResVec<<<1,1>>>(tC, fC, nblockrows);
//    printf("bsrbmv32 nnz in vec: %d\n", countNnzinVec<float>(result_bsrbmv32, nrows));
#endif

    //============================================= check result
    printf("BSR BMV-32-bin: %.3lf\n", bmvbin32_time);

    //============================================= free memory
    // free cusparse bsr metadata
    hipsparseDestroyMatDescr(csr_descr);
    hipsparseDestroyMatDescr(bsr_descr);
    hipsparseDestroy(handle);

    // free mem
    free(h_csrRowPtr);
    free(h_csrColInd);
    free(h_csrVal);

    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);
}

int main(int argc, char* argv[])
{
    main32(argc, argv);
}
